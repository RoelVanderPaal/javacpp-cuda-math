
#include <hip/hip_runtime.h>
extern "C"
__global__ void math_acos(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = acos(x[id]);
    }
}

extern "C"
__global__ void math_acosh(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = acosh(x[id]);
    }
}

extern "C"
__global__ void math_asin(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = asin(x[id]);
    }
}

extern "C"
__global__ void math_asinh(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = asinh(x[id]);
    }
}

extern "C"
__global__ void math_atan(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = atan(x[id]);
    }
}

extern "C"
__global__ void math_atanh(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = atanh(x[id]);
    }
}

extern "C"
__global__ void math_cbrt(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = cbrt(x[id]);
    }
}

extern "C"
__global__ void math_ceil(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = ceil(x[id]);
    }
}

extern "C"
__global__ void math_cos(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = cos(x[id]);
    }
}

extern "C"
__global__ void math_cosh(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = cosh(x[id]);
    }
}

extern "C"
__global__ void math_cospi(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = cospi(x[id]);
    }
}

extern "C"
__global__ void math_cyl_bessel_i0(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = cyl_bessel_i0(x[id]);
    }
}

extern "C"
__global__ void math_cyl_bessel_i1(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = cyl_bessel_i1(x[id]);
    }
}

extern "C"
__global__ void math_erf(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = erf(x[id]);
    }
}

extern "C"
__global__ void math_erfc(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = erfc(x[id]);
    }
}

extern "C"
__global__ void math_erfcinv(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = erfcinv(x[id]);
    }
}

extern "C"
__global__ void math_erfcx(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = erfcx(x[id]);
    }
}

extern "C"
__global__ void math_erfinv(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = erfinv(x[id]);
    }
}

extern "C"
__global__ void math_exp(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = exp(x[id]);
    }
}

extern "C"
__global__ void math_exp10(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = exp10(x[id]);
    }
}

extern "C"
__global__ void math_exp2(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = exp2(x[id]);
    }
}

extern "C"
__global__ void math_expm1(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = expm1(x[id]);
    }
}

extern "C"
__global__ void math_fabs(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = fabs(x[id]);
    }
}

extern "C"
__global__ void math_floor(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = floor(x[id]);
    }
}

extern "C"
__global__ void math_j0(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = j0(x[id]);
    }
}

extern "C"
__global__ void math_j1(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = j1(x[id]);
    }
}

extern "C"
__global__ void math_lgamma(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = lgamma(x[id]);
    }
}

extern "C"
__global__ void math_log(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = log(x[id]);
    }
}

extern "C"
__global__ void math_log10(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = log10(x[id]);
    }
}

extern "C"
__global__ void math_log1p(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = log1p(x[id]);
    }
}

extern "C"
__global__ void math_log2(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = log2(x[id]);
    }
}

extern "C"
__global__ void math_logb(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = logb(x[id]);
    }
}

extern "C"
__global__ void math_nearbyint(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = nearbyint(x[id]);
    }
}

extern "C"
__global__ void math_normcdf(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = normcdf(x[id]);
    }
}

extern "C"
__global__ void math_normcdfinv(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = normcdfinv(x[id]);
    }
}

extern "C"
__global__ void math_rcbrt(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = rcbrt(x[id]);
    }
}

extern "C"
__global__ void math_rint(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = rint(x[id]);
    }
}

extern "C"
__global__ void math_round(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = round(x[id]);
    }
}

extern "C"
__global__ void math_rsqrt(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = rsqrt(x[id]);
    }
}

extern "C"
__global__ void math_sin(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = sin(x[id]);
    }
}

extern "C"
__global__ void math_sinh(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = sinh(x[id]);
    }
}

extern "C"
__global__ void math_sinpi(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = sinpi(x[id]);
    }
}

extern "C"
__global__ void math_sqrt(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = sqrt(x[id]);
    }
}

extern "C"
__global__ void math_tan(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = tan(x[id]);
    }
}

extern "C"
__global__ void math_tanh(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = tanh(x[id]);
    }
}

extern "C"
__global__ void math_tgamma(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = tgamma(x[id]);
    }
}

extern "C"
__global__ void math_trunc(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = trunc(x[id]);
    }
}

extern "C"
__global__ void math_y0(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = y0(x[id]);
    }
}

extern "C"
__global__ void math_y1(size_t n, double *result, double  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = y1(x[id]);
    }
}

