
#include <hip/hip_runtime.h>
extern "C"
__global__ void math_acosf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = acosf(x[id]);
    }
}

extern "C"
__global__ void math_acoshf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = acoshf(x[id]);
    }
}

extern "C"
__global__ void math_asinf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = asinf(x[id]);
    }
}

extern "C"
__global__ void math_asinhf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = asinhf(x[id]);
    }
}

extern "C"
__global__ void math_atanf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = atanf(x[id]);
    }
}

extern "C"
__global__ void math_atanhf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = atanhf(x[id]);
    }
}

extern "C"
__global__ void math_cbrtf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = cbrtf(x[id]);
    }
}

extern "C"
__global__ void math_ceilf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = ceilf(x[id]);
    }
}

extern "C"
__global__ void math_cosf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = cosf(x[id]);
    }
}

extern "C"
__global__ void math_coshf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = coshf(x[id]);
    }
}

extern "C"
__global__ void math_cospif(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = cospif(x[id]);
    }
}

extern "C"
__global__ void math_cyl_bessel_i0f(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = cyl_bessel_i0f(x[id]);
    }
}

extern "C"
__global__ void math_cyl_bessel_i1f(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = cyl_bessel_i1f(x[id]);
    }
}

extern "C"
__global__ void math_erfcf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = erfcf(x[id]);
    }
}

extern "C"
__global__ void math_erfcinvf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = erfcinvf(x[id]);
    }
}

extern "C"
__global__ void math_erfcxf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = erfcxf(x[id]);
    }
}

extern "C"
__global__ void math_erff(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = erff(x[id]);
    }
}

extern "C"
__global__ void math_erfinvf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = erfinvf(x[id]);
    }
}

extern "C"
__global__ void math_exp10f(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = exp10f(x[id]);
    }
}

extern "C"
__global__ void math_exp2f(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = exp2f(x[id]);
    }
}

extern "C"
__global__ void math_expf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = expf(x[id]);
    }
}

extern "C"
__global__ void math_expm1f(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = expm1f(x[id]);
    }
}

extern "C"
__global__ void math_fabsf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = fabsf(x[id]);
    }
}

extern "C"
__global__ void math_floorf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = floorf(x[id]);
    }
}

extern "C"
__global__ void math_j0f(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = j0f(x[id]);
    }
}

extern "C"
__global__ void math_j1f(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = j1f(x[id]);
    }
}

extern "C"
__global__ void math_lgammaf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = lgammaf(x[id]);
    }
}

extern "C"
__global__ void math_log10f(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = log10f(x[id]);
    }
}

extern "C"
__global__ void math_log1pf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = log1pf(x[id]);
    }
}

extern "C"
__global__ void math_log2f(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = log2f(x[id]);
    }
}

extern "C"
__global__ void math_logbf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = logbf(x[id]);
    }
}

extern "C"
__global__ void math_logf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = logf(x[id]);
    }
}

extern "C"
__global__ void math_nearbyintf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = nearbyintf(x[id]);
    }
}

extern "C"
__global__ void math_normcdff(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = normcdff(x[id]);
    }
}

extern "C"
__global__ void math_normcdfinvf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = normcdfinvf(x[id]);
    }
}

extern "C"
__global__ void math_rcbrtf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = rcbrtf(x[id]);
    }
}

extern "C"
__global__ void math_rintf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = rintf(x[id]);
    }
}

extern "C"
__global__ void math_roundf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = roundf(x[id]);
    }
}

extern "C"
__global__ void math_rsqrtf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = rsqrtf(x[id]);
    }
}

extern "C"
__global__ void math_sinf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = sinf(x[id]);
    }
}

extern "C"
__global__ void math_sinhf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = sinhf(x[id]);
    }
}

extern "C"
__global__ void math_sinpif(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = sinpif(x[id]);
    }
}

extern "C"
__global__ void math_sqrtf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = sqrtf(x[id]);
    }
}

extern "C"
__global__ void math_tanf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = tanf(x[id]);
    }
}

extern "C"
__global__ void math_tanhf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = tanhf(x[id]);
    }
}

extern "C"
__global__ void math_tgammaf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = tgammaf(x[id]);
    }
}

extern "C"
__global__ void math_truncf(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = truncf(x[id]);
    }
}

extern "C"
__global__ void math_y0f(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = y0f(x[id]);
    }
}

extern "C"
__global__ void math_y1f(size_t n, float *result, float  *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = y1f(x[id]);
    }
}

